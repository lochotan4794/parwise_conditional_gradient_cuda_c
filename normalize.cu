#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <random>
#include "BPCG.h"


void normalize(float** objects, float** normalizedObjs, int numCoords, int numObjs)
{
    float mean[3] = { 0 };// = (float* ) malloc(numCoords * sizeof(float));
    float std[3] = { 0 };// = (float*)malloc(numCoords * sizeof(float));

    int i, j;

    for (i = 0; i < numObjs; i++)
    {
        for (j = 0; j < numCoords; j++)
        {
            mean[j] = mean[j] + objects[i][j];
        }
    }

    for (j = 0; j < numCoords; j++)
    {
        mean[j] = mean[j] / numObjs;
    }

    for (i = 0; i < numObjs; i++)
    {
        for (j = 0; j < numCoords; j++)
        {
            std[j] = std[j] + (objects[i][j] - mean[j]) * (objects[i][j] - mean[j]);
        }
    }

    for (j = 0; j < numCoords; j++)
    {
        std[j] =  sqrt(std[j] / numObjs);
    }


    for (i = 0; i < numObjs; i++)
    {
        for (j = 0; j < numCoords; j++)
        {
            normalizedObjs[i][j] = (objects[i][j] - mean[j]) / std[j];
        }
    }

}